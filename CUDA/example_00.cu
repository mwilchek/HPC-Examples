#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void AddIntsCUDA (int *a, int *b)
{
a[0]+=b[0];
}

int main()
{
  int a = 5, b = 9;
  int *d_a, *d_b ;

  hipMalloc(&d_a,sizeof(int));
  hipMalloc(&d_b,sizeof(int));

  hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);

  AddIntsCUDA<<<1, 1>>>(d_a,d_b);


  hipMemcpy(&a,d_a,sizeof(int),hipMemcpyDeviceToHost);
  cout <<"The answer is "<<a<<endl;
  return 0;
}
