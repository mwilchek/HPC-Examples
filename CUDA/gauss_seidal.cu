
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <omp.h>


#define BLOCK_SIZE 32

void print_matrix_sample(double *M,int ROWS, int COLS)
{
	int i, j;
	for (i = 10; i >= 0; i--)
	{
		for (j = 0; j < 10; j++)
		{
			printf("%.2f\t", M[j * COLS + i]);
		}
		printf("\n");
	}
}


void init_grid_U(double * grid, double * grid_prev, int ROWS, int COLS)
{
#pragma omp parallel
        {
        #pragma omp single
                {
                        printf("Number of OMP Threads = %d \n",omp_get_num_threads());
                }
        #pragma omp for
                for (int i = 0; i < ROWS; i++) {
                        for (int j = 0; j < COLS; j++) {
                                grid    [i * (COLS) + j] = 0.0;
                                grid_prev[i * (COLS) + j] = 0.0;

                                // --- Boundary condition on the left, right and upper walls. The boundary condition on the lower wall is vanishing temperature
                                if (i == (ROWS - 1) || i == 0 || j == (COLS - 1))
                                {
                                        grid    [i * (COLS) + j] = 20.0;
                                        grid_prev[i * (COLS) + j] = 20.0;
                                }
                        }
                }
        }
}


__global__ void compute_gridpoints_GPU(double * grid, double * grid_prev, const int ROWS, const int COLS)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < ROWS - 1 && j < COLS - 1 && i > 0 && j > 0)
	{
		grid[j * COLS + i] = (grid[j * COLS + (i - 1)] + grid[j * COLS + (i + 1)] + grid[(j - 1) * COLS + i] + grid[(j + 1) * COLS + i] ) * 0.25;

	}
}


int main()
{

	const int ROWS         = 10000+2;                           // --- Grid size is N x N
	const int COLS         = 10000+2;                           // --- Grid size is N x N
	const int MAX_ITER  = 1000;                         // --- Maximum number of iterations

	struct timeval start_time_CPU, stop_time_CPU, elapsed_time_CPU;
	struct timeval start_time_GPU, stop_time_GPU, elapsed_time_GPU;



	// --- Allocating host memory variables
	double *h_grid             = (double *)malloc( (ROWS) * (COLS) * sizeof(double));
	double *h_grid_prev         = (double *)malloc( (ROWS) * (COLS) * sizeof(double));

	// --- Allocating device memory variables
	double *d_grid;            
	hipMalloc(&d_grid, (ROWS) * (COLS) * sizeof(double));
	double *d_grid_prev;        
	hipMalloc(&d_grid_prev, (ROWS) * (COLS) * sizeof(double));

	// --- Host array initialization

        gettimeofday(&start_time_CPU,NULL);
	init_grid_U(h_grid, h_grid_prev, ROWS, COLS);
        gettimeofday(&stop_time_CPU,NULL);
	print_matrix_sample(h_grid,ROWS,COLS);
	hipMemcpy(d_grid,       h_grid,        ROWS * COLS * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_grid_prev,       h_grid_prev,        ROWS * COLS * sizeof(double), hipMemcpyHostToDevice);




	// --- Device iterations
	dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 DimGrid((COLS+DimBlock.x-1)/DimBlock.x,(ROWS+DimBlock.y-1)/DimBlock.y);
	printf("DimBlock.x = %d DimBlock.y = %d\n",DimBlock.x,DimBlock.y);
	printf("DimGrid.x = %d DimGrid.y = %d\n",DimGrid.x,DimGrid.y);


        gettimeofday(&start_time_GPU,NULL);
	for (int h = 0; h < MAX_ITER; h++)
	{
		compute_gridpoints_GPU<<<DimGrid, DimBlock>>>(d_grid, d_grid_prev, ROWS,COLS);

		// --- Pointers swap
		//temp = d_grid;
		//d_grid = d_grid_prev;
		//d_grid_prev = temp;
	}
        gettimeofday(&stop_time_GPU,NULL);

	hipMemcpy(h_grid, d_grid, (ROWS) * (COLS) * sizeof(double), hipMemcpyDeviceToHost);
	print_matrix_sample(h_grid,ROWS,COLS);
        timersub(&stop_time_CPU, &start_time_CPU, &elapsed_time_CPU);
        timersub(&stop_time_GPU, &start_time_GPU, &elapsed_time_GPU);

        printf("CPU Time: %f   GPU Time: %f \n", elapsed_time_CPU.tv_sec+elapsed_time_CPU.tv_usec/1000000.0, elapsed_time_GPU.tv_sec+elapsed_time_GPU.tv_usec/1000000.0);

	free(h_grid);
	free(h_grid_prev);
	hipFree(d_grid);
	hipFree(d_grid_prev);

	return 0;
}
